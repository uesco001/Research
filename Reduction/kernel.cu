#include "hip/hip_runtime.h"

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE
#define BLOCK_SIZE 512
__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE


	float sdata[];
	unsigned i = threadIdx.x + blockIdx.x * blockDim.x;
	// stride is total number of threadsi
	unsigned tid = threadIdx.x;
	// All threads handle blockDim.x * gridDim.x
	// consecutive elements
	sdata[tid] = in[i];
	_syncthreads();
	
	for( unsigned s = 1; s < blockDim.x; s *=2)
	{
		if(tid%(2*s) == 0)
		{
			sdatat[tid] += sdata[tid + s];
		}
		_synchthreads();
	}
	
	if(tid == 0){ out[blockIdx.x] = sdata[0];}

}
